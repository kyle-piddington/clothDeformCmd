#include "hip/hip_runtime.h"

#include "ClothForceIntegrator.h"
#include <algorithm>
#include <iostream>

#define YOUNG_MOD 1000.0 //N/m
#define POISSON_COEFF 0.85
#define MASS 600.0 //kg/m
#define GRAVITY -4.5
#define DAMPN  0.05
#define COLLISIONSTR 20.0

#define MIN_STEP 0.0005 //Seconds

const double YoungPoissonMatrixScalar = YOUNG_MOD/(1 - POISSON_COEFF * POISSON_COEFF);

struct Vector
{
   double x;
   double y;
   double z;
};

struct Vector sumPoint(Vector & a, Vector & b, Vector & c,
						  double  wUA, double  wUB, double  wUC)
{

   struct Vector point;
   point.x = a.x * wUA + b.x * wUB + c.x * wUC;
   point.y = a.y * wUA + b.y * wUB + c.y * wUC;
   point.z = a.z * wUA + b.z * wUB + c.z * wUC;
   return point;
}

Vector calculateForce(Vector U, Vector V,
							 double rUJ, double rVJ, double d)
{
   double euu = 0.5 * (U.x * U.x + U.y * U.y + U.z * U.z -1);
   double evv = 0.5 * (V.x * V.x + V.y * V.y + V.z * V.z -1);
   //EUV is 0.5* (UTV + VTU), should be 2 UtV
   double euv =  U.x * V.x + U.y * V.y + U.z * V.z;

   Vector sigmas;
   sigmas.x = euu + POISSON_COEFF * evv ;
   sigmas.y = POISSON_COEFF*evv + euu ;
   sigmas.z = euv * (1 - POISSON_COEFF) / 2 ;

   sigmas.x *= YoungPoissonMatrixScalar;
   sigmas.y *= YoungPoissonMatrixScalar;
   sigmas.z *= YoungPoissonMatrixScalar;
   Vector force;
   force.x = -fabs(d)/2 *(
			   sigmas.x * rUJ * U.x +
			   sigmas.y * rVJ * V.x +
			   sigmas.z * (rUJ * V.x + rVJ * U.x));

   force.y = -fabs(d)/2 * (
			   sigmas.x * rUJ * U.y +
			   sigmas.y * rVJ * V.y +
			   sigmas.z * (rUJ * V.y + rVJ * U.y));

   force.z = -fabs(d)/2 * (
			   sigmas.x * rUJ * U.z +
			   sigmas.y * rVJ * V.z +
			   sigmas.z * (rUJ * V.z + rVJ * U.z)) ;
   return force;
}

/**
 * Setup triangle weights organized by indicies
 */
inline void ClothForceIntegrator::caluclateTriangleWeights(std::vector<float> & weights,std::vector<int> & inds)
{
   /**
	* Iterate through each triangle and calcluate a weight grouping
	*/
   for(int i = 0; i < numTriangles*3; i+=3)
   {
     floatVec2D a, b, c;
     a.x = weights[2*inds[i]];
     a.y = weights[2*inds[i]+1];
     b.x = weights[2*inds[i+1]];
     b.y = weights[2*inds[i+1]+1];
     c.x = weights[2*inds[i+2]];
     c.y = weights[2*inds[i+2]+1];
	  //create and set weights
	  float d = a.x * (b.y - c.y) + b.x * (c.y - a.y) + c.x * (a.y - b.y);
	  float recrip = 1.0 /  d;
	  dArray[i/3] = d;
	  wUA[i/3] = (b.y - c.y) * recrip;
	  wVA[i/3] = (c.x - b.x) * recrip;
	  wUB[i/3] = (c.y - a.y) * recrip;
	  wVB[i/3] = (a.x - c.x) * recrip;
	  wUC[i/3] = (a.y - b.y) * recrip;
	  wVC[i/3] = (b.x - a.x) * recrip;
   }
}

inline double lengthSq(Vector v)
{
   return v.x * v.x + v.y * v.y + v.z * v.z;
}

/**
 * Create the initial arrays, and organize data in SOA structure, push to the Phi.
 * @param cloth [description]
 */
 /**
   OrigIndicies:
   [a b c] [a b c]
   vertices:
   [x y z] [ x y z] [x y z] */
ClothForceIntegrator::ClothForceIntegrator(std::vector<int>  orig_indices, std::vector<float>  vertices, std::vector<float>  weights):
	numTriangles(orig_indices.size()/3),
	numVerts(vertices.size()/3),
	numIndicies(orig_indices.size())

{
   /**
	* Replace with __mm_malloc later
	*/
   
   std::cout << "numVerts: " << numVerts << std::endl;
   
   indicies = new int[orig_indices.size()];
   wUA = new double[numTriangles];
   wUB = new double[numTriangles];
   wUC = new double[numTriangles];
   wVA = new double[numTriangles];
   wVB = new double[numTriangles];
   wVC = new double[numTriangles];
   dArray = new double[numTriangles];

   vertsX = new double[numVerts];
   vertsY = new double[numVerts];
   vertsZ = new double[numVerts];
   velsX = new double[numVerts];
   velsY = new double[numVerts];
   velsZ = new double[numVerts];

   forceX = new double[numVerts];
   forceY = new double[numVerts];
   forceZ = new double[numVerts];
   
   expandedForceX = new double[numTriangles*3];
   expandedForceY = new double[numTriangles*3];
   expandedForceZ = new double[numTriangles*3];

   for(int i = 0; i < numVerts; i++)
   {
	  vertsX[i] = vertices[3*i];
	  vertsY[i] = vertices[3*i+1];
	  vertsZ[i] = vertices[3*i+2];
   }

   std::fill(velsX, velsX + numVerts, 0);
   std::fill(velsY, velsY + numVerts, 0);
   std::fill(velsZ, velsZ + numVerts, 0);

   std::fill(forceX, forceX + numVerts, 0);
   std::fill(forceY, forceY + numVerts, 0);
   std::fill(forceZ, forceZ + numVerts, 0);

   for(int i = 0; i < numIndicies; i++)
   {
      indicies[i] = orig_indices[i];
   }
   caluclateTriangleWeights(weights,orig_indices);
   
   
   counts = new unsigned int[numVerts];
   locs = new unsigned int[numVerts];
   outIdx = new unsigned int[numTriangles*3];
   
   for(int i = 0; i < numVerts; i++) { 	// init counts
      counts[i] = 0;
   }
   
   for(int i = 0; i < numIndicies; i++) {
      counts[indicies[i]]++;
   }
   
   std::cout << "init'd counts" << std::endl;
   
   locs[0] = 0;   	// init locs
   for(int i = 1; i < numVerts; i++) {
      locs[i] = counts[i - 1] + locs[i - 1];
   }
   
   unsigned int *tempCounts = new unsigned int[numVerts];   	// init outIdx
   for(int i = 0; i < numVerts; i++) {
      tempCounts[i] = 0;
   }
   
   for(int i = 0; i < numIndicies; i++) {
      int vertex = indicies[i];
      outIdx[i] = locs[vertex] + tempCounts[vertex];
      tempCounts[vertex]++;
   }
   
   std::cout << "dumping counts: ";
   for (int i = 0; i < numVerts; i++) {
      std::cout << counts[i] << " ";
   }
   std::cout << std::endl;
   
   std::cout << "dumping locs: ";
   for (int i = 0; i < numVerts; i++) {
      std::cout << locs[i] << " ";
   }
   std::cout << std::endl;
   
   std::cout << "dumping outIdx: ";
   for (int i = 0; i < numIndicies; i++) {
      std::cout << outIdx[i] << " ";
   }
   std::cout << std::endl;
   
    std::cout << "dumping indicies: ";
   for (int i = 0; i < numIndicies; i++) {
      std::cout << indicies[i] << " ";
   }
   std::cout << std::endl;
   
   std::cout << "ran init without dying" << std::endl;
//   exit(EXIT_FAILURE);
}

void ClothForceIntegrator::step(double stepAmnt, float * outputVertices, std::vector<int> & theLockedVerts )
{
   //Calculate a force vector
   int numLockedVerts = theLockedVerts.size();
   int *lockedVerts = new int[numLockedVerts];
   for(int i = 0; i < theLockedVerts.size(); i++)
   {
      lockedVerts[i] = theLockedVerts[i];
   }

   for(int steps = 0; steps < (int)(stepAmnt/MIN_STEP); steps++)
   {
    /*  std::cout << "dumping counts: ";
      for (int i = 0; i < numVerts; i++) {
         std::cout << counts[i] << " ";
      }
      std::cout << std::endl;
      
      std::cout << "dumping locs: ";
      for (int i = 0; i < numVerts; i++) {
         std::cout << locs[i] << " ";
      }
      std::cout << std::endl;
      
      std::cout << "dumping outIdx: ";
      for (int i = 0; i < numIndicies; i++) {
         std::cout << outIdx[i] << " ";
      }
      std::cout << std::endl;
      
       std::cout << "dumping indicies: ";
      for (int i = 0; i < numIndicies; i++) {
         std::cout << indicies[i] << " ";
      }
 
      std::cout << std::endl;
      std::cout << "dumping vertsX: ";
      for (int i = 0; i < numVerts; i++) {
         std::cout << vertsX[i] << " ";
      }
      std::cout << std::endl;
 
      std::cout << "dumping vertsY: ";
      for (int i = 0; i < numVerts; i++) {
         std::cout << vertsY[i] << " ";
      }
       std::cout << "dumping vertsZ: ";
      for (int i = 0; i < numVerts; i++) {
         std::cout << vertsZ[i] << " ";
      }
      std::cout << std::endl;*/
   
      const double dt = MIN_STEP;
      const double recipMass = 1/(MASS/numVerts);

	   for(int i = 0; i < numTriangles; i++)
	   {
		  Vector A, B, C, vA, vB, vC;

		  A.x = vertsX[indicies[i*3]];
		  B.x = vertsX[indicies[i*3 + 1]];
		  C.x = vertsX[indicies[i*3 + 2]];

		  A.y = vertsY[indicies[i*3]];
		  B.y = vertsY[indicies[i*3 + 1]];
		  C.y = vertsY[indicies[i*3 + 2]];

		  A.z = vertsZ[indicies[i*3]];
		  B.z = vertsZ[indicies[i*3 + 1]];
		  C.z = vertsZ[indicies[i*3 + 2]];

		  vA.x = velsX[indicies[i*3]];
		  vB.x = velsX[indicies[i*3 + 1]];
		  vC.x = velsX[indicies[i*3 + 2]];

		  vA.y = velsY[indicies[i*3]];
		  vB.y = velsY[indicies[i*3 + 1]];
		  vC.y = velsY[indicies[i*3 + 2]];

		  vA.z = velsZ[indicies[i*3]];
		  vB.z = velsZ[indicies[i*3 + 1]];
		  vC.z = velsZ[indicies[i*3 + 2]];

		  Vector U = sumPoint(A,B,C,wUA[i],wUB[i],wUC[i]);
		  Vector V = sumPoint(A,B,C,wVA[i],wVB[i],wVC[i]);

		  Vector forceA = calculateForce(U,V,wUA[i],wVA[i],dArray[i]);
		  Vector forceB = calculateForce(U,V,wUB[i],wVB[i],dArray[i]);
		  Vector forceC = calculateForce(U,V,wUC[i],wVC[i],dArray[i]);

		  //Update first vertex
		  expandedForceX[outIdx[i*3]] = forceA.x - DAMPN * vA.x;
		  expandedForceY[outIdx[i*3]] = forceA.y - DAMPN * vA.y;
		  expandedForceZ[outIdx[i*3]] = forceA.z - DAMPN * vA.z;

		  //Update second vertex
		  expandedForceX[outIdx[i*3+1]] = forceB.x - DAMPN * vB.x;
		  expandedForceY[outIdx[i*3+1]] = forceB.y - DAMPN * vB.y;
		  expandedForceZ[outIdx[i*3+1]] = forceB.z - DAMPN * vB.z;

		  //update third vertex
		  expandedForceX[outIdx[i*3 + 2]] = forceC.x - DAMPN * vC.x;
		  expandedForceY[outIdx[i*3 + 2]] = forceC.y - DAMPN * vC.y;
		  expandedForceZ[outIdx[i*3 + 2]] = forceC.z - DAMPN * vC.z;
	   }
      std::cout << "dumping ExpandedForceX: ";
      for (int i = 0; i < numTriangles*3; i++) {
         std::cout << expandedForceX[i] << " ";
      }
       std::cout << "dumping ExpandedForceY: ";
      for (int i = 0; i < numTriangles*3; i++) {
         std::cout << expandedForceY[i] << " ";
      }
       std::cout << "dumping ExpandedForceZ: ";
      for (int i = 0; i < numTriangles*3; i++) {
         std::cout << expandedForceZ[i] << " ";
      }
      std::cout << std::endl;
   
      // todo: calc forceXYZ with expandedForceXYZ
      for (int i = 0; i < numTriangles*3; i++) {
         unsigned int theLoc = locs[indicies[i]];
         
//         if (theLoc <= numIndicies) {
            forceX[theLoc] += expandedForceX[i];
            forceY[theLoc] += expandedForceY[i];
            forceZ[theLoc] += expandedForceZ[i];
//         }
//         else {
//            std::cout << "i: " << i << ", theLoc: " << theLoc << std::endl;
//         }
      }

	   for(int i = 0; i < numVerts; i++)
	   {
		  forceY[i] += GRAVITY;
	   }
      
// 	   for(int i = 0; i < numLockedVerts; ++i)
// 	   {
// 		  forceX[lockedVerts[i]]=0;
// 		  forceY[lockedVerts[i]]=0;
// 		  forceZ[lockedVerts[i]]=0;
// 	   }

   	   for(int i = 0; i < numVerts; i++)
   	   {
   		  velsX[i]  += forceX[i] * dt * recipMass;
   		  vertsX[i] += velsX[i] * dt;

   		  velsY[i]  += forceY[i] * dt * recipMass;
   		  vertsY[i] += velsY[i] * dt;

   		  velsZ[i]  += forceZ[i] * dt * recipMass;
   		  vertsZ[i] += velsZ[i] * dt;
   	   }

      for(int i = 0; i < numVerts; i++) {
         forceX[i] = forceY[i] = forceZ[i] = 0;
      }
   }
   delete [] lockedVerts;
   /**
	* Set final vertex positions
	*/
   for(int i = 0; i < numVerts; i++)
   {
	  outputVertices[i*3] = (float)vertsX[i];
	  outputVertices[i*3+1] = (float)vertsY[i];
	  outputVertices[i*3+2] = (float)vertsZ[i];  
   }
    std::cout << "dumping outputVertices: ";
      for (int i = 0; i < numVerts*3; i++) {
         std::cout << outputVertices[i] << " ";
      }
      std::cout << std::endl;
   
}

ClothForceIntegrator::~ClothForceIntegrator()
{
   delete [] wUA ;
   delete [] wUB ;
   delete [] wUC ;
   delete [] wVA ;
   delete [] wVB ;
   delete [] wVC ;
   delete [] vertsX;
   delete [] vertsY;
   delete [] vertsZ;
   delete [] velsX;
   delete [] velsY;
   delete [] velsZ;
   delete [] indicies;
   delete [] forceX;
   delete [] forceY;
   delete [] forceZ;
   delete [] dArray;
}
