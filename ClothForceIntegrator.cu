#include "hip/hip_runtime.h"

#include "ClothForceIntegrator.h"
#include <algorithm>
#include <iostream>

#define YOUNG_MOD 1000.0 //N/m
#define POISSON_COEFF 0.85
#define MASS 600.0 //kg/m
#define GRAVITY -4.5
#define DAMPN  0.05
#define COLLISIONSTR 20.0

#define ALLOC alloc_if(1)
#define FREE free_if(1)
#define RETAIN free_if(0)
#define REUSE alloc_if(0)
#define MIN_STEP 0.0005 //Seconds


//#include <omp.h>

 const double YoungPoissonMatrixScalar = YOUNG_MOD/(1 - POISSON_COEFF * POISSON_COEFF);

/**
 * Push the force derivative structure to the Phi
 */
struct Vector
{
   double x;
   double y;
   double z;
};

 struct Vector sumPoint(Vector & a, Vector & b, Vector & c,
						  double  wUA, double  wUB, double  wUC)
{

   struct Vector point;
   point.x = a.x * wUA + b.x * wUB + c.x * wUC;
   point.y = a.y * wUA + b.y * wUB + c.y * wUC;
   point.z = a.z * wUA + b.z * wUB + c.z * wUC;
   return point;
}



 Vector calculateForce(Vector U, Vector V,
							 double rUJ, double rVJ, double d)
{
   double euu = 0.5 * (U.x * U.x + U.y * U.y + U.z * U.z -1);
   double evv = 0.5 * (V.x * V.x + V.y * V.y + V.z * V.z -1);
   //EUV is 0.5* (UTV + VTU), should be 2 UtV
   double euv =  U.x * V.x + U.y * V.y + U.z * V.z;




   Vector sigmas;
   sigmas.x = euu + POISSON_COEFF * evv ;
   sigmas.y = POISSON_COEFF*evv + euu ;
   sigmas.z = euv * (1 - POISSON_COEFF) / 2 ;

   sigmas.x *= YoungPoissonMatrixScalar;
   sigmas.y *= YoungPoissonMatrixScalar;
   sigmas.z *= YoungPoissonMatrixScalar;
   Vector force;
   force.x = -fabs(d)/2 *(
			   sigmas.x * rUJ * U.x +
			   sigmas.y * rVJ * V.x +
			   sigmas.z * (rUJ * V.x + rVJ * U.x));

   force.y = -fabs(d)/2 * (
			   sigmas.x * rUJ * U.y +
			   sigmas.y * rVJ * V.y +
			   sigmas.z * (rUJ * V.y + rVJ * U.y));

   force.z = -fabs(d)/2 * (
			   sigmas.x * rUJ * U.z +
			   sigmas.y * rVJ * V.z +
			   sigmas.z * (rUJ * V.z + rVJ * U.z)) ;
   return force;
}


/**
 * Setup triangle weights organized by indicies
 */
inline void ClothForceIntegrator::caluclateTriangleWeights(std::vector<float> & weights,std::vector<int> & inds)
{

   /**
	* Iterate through each triangle and calcluate a weight grouping
	*/
   for(int i = 0; i < numTriangles*3; i+=3)
   {
	  std::cout << i << std::endl;
     floatVec2D a, b, c;
     a.x = weights[2*inds[i]];
     a.y = weights[2*inds[i]+1];
     b.x = weights[2*inds[i+1]];
     b.y = weights[2*inds[i+1]+1];
     c.x = weights[2*inds[i+2]];
     c.y = weights[2*inds[i+2]+1];
	  std::cout << std::endl;
	  //create and set weights
	  float d = a.x * (b.y - c.y) + b.x * (c.y - a.y) + c.x * (a.y - b.y);
	  float recrip = 1.0 /  d;
	  dArray[i/3] = d;
	  wUA[i/3] = (b.y - c.y) * recrip;
	  wVA[i/3] = (c.x - b.x) * recrip;
	  wUB[i/3] = (c.y - a.y) * recrip;
	  wVB[i/3] = (a.x - c.x) * recrip;
	  wUC[i/3] = (a.y - b.y) * recrip;
	  wVC[i/3] = (b.x - a.x) * recrip;
   }

}



inline double lengthSq(Vector v)
{
   return v.x * v.x + v.y * v.y + v.z * v.z;
}


/**
 * Create the initial arrays, and organize data in SOA structure, push to the Phi.
 * @param cloth [description]
 */
void ClothForceIntegrator::init(std::vector<int>  orig_indices, std::vector<float>  vertices, std::vector<float>  weights)
{

   /**
	* Replace with __mm_malloc later
	*/
   numTriangles = orig_indices.size()/3;
   numVerts = vertices.size()/3;
   indicies = new int[orig_indices.size()];
   wUA = new double[numTriangles];
   wUB = new double[numTriangles];
   wUC = new double[numTriangles];
   wVA = new double[numTriangles];
   wVB = new double[numTriangles];
   wVC = new double[numTriangles];
   dArray = new double[numTriangles];

   vertsX = new double[numVerts];
   vertsY = new double[numVerts];
   vertsZ = new double[numVerts];
   velsX = new double[numVerts];
   velsY = new double[numVerts];
   velsZ = new double[numVerts];

   forceX = new double[numVerts];
   forceY = new double[numVerts];
   forceZ = new double[numVerts];

   for(int i = 0; i < numVerts; i++)
   {
	  vertsX[i] = vertices[3*i];
	  vertsY[i] = vertices[3*i+1];
	  vertsZ[i] = vertices[3*i+2];
   }

   std::fill(velsX, velsX + numVerts, 0);
   std::fill(velsY, velsY + numVerts, 0);
   std::fill(velsZ, velsZ + numVerts, 0);

   std::fill(forceX, forceX + numVerts, 0);
   std::fill(forceY, forceY + numVerts, 0);
   std::fill(forceZ, forceZ + numVerts, 0);


   memcpy(indicies,orig_indices.data(), sizeof(int)*orig_indices.size());
   caluclateTriangleWeights(weights,orig_indices);
	#ifdef OFFLOAD
	#pragma offload_transfer target(mic)\
		in(wUA: length(numTriangles) ALLOC RETAIN)\
		in(wUB: length(numTriangles) ALLOC RETAIN)\
		in(wUC: length(numTriangles) ALLOC RETAIN)\
		in(wVA: length(numTriangles) ALLOC RETAIN)\
		in(wVB: length(numTriangles) ALLOC RETAIN)\
		in(wVC: length(numTriangles) ALLOC RETAIN)\
		in(dArray: length(numTriangles) ALLOC RETAIN)\
		in(vertsX: length(numVerts) ALLOC RETAIN)\
		in(vertsY: length(numVerts) ALLOC RETAIN)\
		in(vertsZ: length(numVerts) ALLOC RETAIN)\
		in(velsX: length(numVerts) ALLOC RETAIN)\
		in(velsY: length(numVerts) ALLOC RETAIN)\
		in(velsZ: length(numVerts) ALLOC RETAIN)\
		in(forceX: length(numVerts) ALLOC RETAIN)\
		in(forceY: length(numVerts) ALLOC RETAIN)\
		in(forceZ: length(numVerts) ALLOC RETAIN)\
		in(indicies: length(numTriangles*3) ALLOC RETAIN)
	#endif

}

void ClothForceIntegrator::step(double stepAmnt, float * outputVertices, std::vector<int> & theLockedVerts )
{
   //Calculate a force vector

   int numLockedVerts = theLockedVerts.size();
   int *lockedVerts = new int[numLockedVerts];
   memcpy(lockedVerts, theLockedVerts.data(), numLockedVerts * sizeof(int));

   std::cout << "sup, bitches" << std::endl;
   std::cout << "numVerts: " << numVerts << std::endl;
   std::cout << "outputVertices: " << outputVertices << std::endl;
   std::cout << "vertsX: " << vertsX << std::endl;
   std::cout << "vertsY: " << vertsY << std::endl;
   std::cout << "vertsZ: " << vertsZ << std::endl;
   
   #ifdef OFFLOAD
   #pragma offload target(mic)\
      nocopy(wUA: length(numTriangles) REUSE RETAIN)\
      nocopy(wUB: length(numTriangles) REUSE RETAIN)\
      nocopy(wUC: length(numTriangles) REUSE RETAIN)\
      nocopy(wVA: length(numTriangles) REUSE RETAIN)\
      nocopy(wVB: length(numTriangles) REUSE RETAIN)\
      nocopy(wVC: length(numTriangles) REUSE RETAIN)\
      nocopy(dArray: length(numTriangles) REUSE RETAIN)\
      nocopy(vertsX: length(numVerts) REUSE RETAIN)\
      nocopy(vertsY: length(numVerts) REUSE RETAIN)\
      nocopy(vertsZ: length(numVerts) REUSE RETAIN)\
      nocopy(velsX: length(numVerts) REUSE RETAIN)\
      nocopy(velsY: length(numVerts) REUSE RETAIN)\
      nocopy(velsZ: length(numVerts) REUSE RETAIN)\
      nocopy(forceX: length(numVerts) REUSE RETAIN)\
      nocopy(forceY: length(numVerts) REUSE RETAIN)\
      nocopy(forceZ: length(numVerts) REUSE RETAIN)\
      nocopy(indicies: length(numTriangles*3) REUSE RETAIN)\
      in(stepAmnt)\
      in(numTriangles)\
      in(numVerts)\
      in(lockedVerts: length(numLockedVerts))\
      in(numLockedVerts)
   #endif
  
   for(int steps = 0; steps < (int)(stepAmnt/MIN_STEP); steps++)
   {
      // std::cout << "offloaded some stuff" << std::endl;
      printf(" offloaded some stuff\n");

      const double dt = MIN_STEP;
      const double recipMass = 1/(MASS/numVerts);

	   #pragma omp parallel for
	   for(int i = 0; i < numTriangles; i++)
	   {
		  Vector A, B, C, vA, vB, vC;

		  A.x = vertsX[indicies[i*3]];
		  B.x = vertsX[indicies[i*3 + 1]];
		  C.x = vertsX[indicies[i*3 + 2]];

		  A.y = vertsY[indicies[i*3]];
		  B.y = vertsY[indicies[i*3 + 1]];
		  C.y = vertsY[indicies[i*3 + 2]];

		  A.z = vertsZ[indicies[i*3]];
		  B.z = vertsZ[indicies[i*3 + 1]];
		  C.z = vertsZ[indicies[i*3 + 2]];

		  vA.x = velsX[indicies[i*3]];
		  vB.x = velsX[indicies[i*3 + 1]];
		  vC.x = velsX[indicies[i*3 + 2]];

		  vA.y = velsY[indicies[i*3]];
		  vB.y = velsY[indicies[i*3 + 1]];
		  vC.y = velsY[indicies[i*3 + 2]];

		  vA.z = velsZ[indicies[i*3]];
		  vB.z = velsZ[indicies[i*3 + 1]];
		  vC.z = velsZ[indicies[i*3 + 2]];

		  Vector U = sumPoint(A,B,C,wUA[i],wUB[i],wUC[i]);
		  Vector V = sumPoint(A,B,C,wVA[i],wVB[i],wVC[i]);

		  Vector forceA = calculateForce(U,V,wUA[i],wVA[i],dArray[i]);
		  Vector forceB = calculateForce(U,V,wUB[i],wVB[i],dArray[i]);
		  Vector forceC = calculateForce(U,V,wUC[i],wVC[i],dArray[i]);

		  //Update first vertex
		  #pragma omp atomic
		  forceX[indicies[i*3]] += forceA.x - DAMPN * vA.x;
		  #pragma omp atomic
		  forceY[indicies[i*3]] += forceA.y - DAMPN * vA.y;
		  #pragma omp atomic
		  forceZ[indicies[i*3]] += forceA.z - DAMPN * vA.z;

		  //Update second vertex
		  #pragma omp atomic
		  forceX[indicies[i*3+1]] += forceB.x - DAMPN * vB.x;
		  #pragma omp atomic     
		  forceY[indicies[i*3+1]] += forceB.y - DAMPN * vB.y;
		  #pragma omp atomic
		  forceZ[indicies[i*3+1]] += forceB.z - DAMPN * vB.z;

		  //update third vertex
		  #pragma omp atomic
		  forceX[indicies[i*3 + 2]] += forceC.x - DAMPN * vC.x;
		  #pragma omp atomic
		  forceY[indicies[i*3 + 2]] += forceC.y - DAMPN * vC.y;
		  #pragma omp atomic
		  forceZ[indicies[i*3 + 2]] += forceC.z - DAMPN * vC.z;
	   }


	   for(int i = 0; i < numVerts; i++)
	   {
		  forceY[i] += GRAVITY;
	   }
	   for(int i = 0; i < numLockedVerts; ++i)
	   {
		  forceX[lockedVerts[i]]=0;
		  forceY[lockedVerts[i]]=0;
		  forceZ[lockedVerts[i]]=0;

	   }

	   for(int i = 0; i < numVerts; i++)
	   {
		  velsX[i]  += forceX[i] * dt * recipMass;
		  vertsX[i] += velsX[i] * dt;

		  velsY[i]  += forceY[i] * dt * recipMass;
		  vertsY[i] += velsY[i] * dt;

		  velsZ[i]  += forceZ[i] * dt * recipMass;
		  vertsZ[i] += velsZ[i] * dt;
	   }

      for(int i = 0; i < numVerts; i++) {
         forceX[i] = forceY[i] = forceZ[i] = 0;
      }
	   
	   // memset(forceX, numVerts * sizeof(double), 0);
	   // memset(forceY, numVerts * sizeof(double), 0);
	   // memset(forceZ, numVerts * sizeof(double), 0);
   }

   std::cout << "didn't die" << std::endl;
   std::cout << "numVerts: " << numVerts << std::endl;
   std::cout << "outputVertices: " << outputVertices << std::endl;
   std::cout << "vertsX: " << vertsX << std::endl;
   std::cout << "vertsY: " << vertsY << std::endl;
   std::cout << "vertsZ: " << vertsZ << std::endl;
   /**
	* Set final vertex positions
	*/
  
   for(int i = 0; i < numVerts; i++)
   {
	  outputVertices[i*3] = (float)vertsX[i];
	  outputVertices[i*3+1] = (float)vertsY[i];
	  outputVertices[i*3+2] = (float)vertsZ[i];  
   }
 
   std::cout << "didn't die the sequel" << std::endl;  

   //Print data for debugging.  But FP error...
   /*
   if(time < 2.0){
	  //printf("%f ", time);
	  for(int i = 0; i < numVerts; i++)
	  {
		 printf("(x:%f y:%f z:%f),", outputVertices[i*3], outputVertices[i*3+1], outputVertices[i*3+2]);
	  }
	  printf("\n\n");
   }

   */



   //Write to output vertices

   
}
void ClothForceIntegrator::startOffload()
{
   
   #ifdef OFFLOAD
   #pragma offload_transfer target(mic)\
		in(vertsX: length(numVerts) REUSE RETAIN)\
		in(vertsY: length(numVerts) REUSE RETAIN)\
		in(vertsZ: length(numVerts) REUSE RETAIN)\
		in(velsX: length(numVerts) REUSE RETAIN)\
		in(velsY: length(numVerts) REUSE RETAIN)\
		in(velsZ: length(numVerts) REUSE RETAIN)
	#endif
	#ifndef OFFLOAD
	  std::cout << "startOffload: Not compiled with Offload, please recompile" << std::endl;
	#endif
}
void ClothForceIntegrator::endOffload()
{

	#ifdef OFFLOAD
	#pragma offload_transfer target(mic)\
		out(vertsX: length(numVerts) REUSE RETAIN)\
		out(vertsY: length(numVerts) REUSE RETAIN)\
		out(vertsZ: length(numVerts) REUSE RETAIN)\
		out(velsX: length(numVerts) REUSE RETAIN)\
		out(velsY: length(numVerts) REUSE RETAIN)\
		out(velsZ: length(numVerts) REUSE RETAIN)
	#endif
	#ifndef OFFLOAD
	  std::cout << "endOffload: Not compiled with Offload, please recompile" << std::endl;
	#endif
}
ClothForceIntegrator::~ClothForceIntegrator()
{
   
	#ifdef OFFLOAD
	#pragma offload_transfer target(mic)\
		nocopy(wUA: length(numTriangles) FREE)\
		nocopy(wUB: length(numTriangles) FREE)\
		nocopy(wUC: length(numTriangles) FREE)\
		nocopy(wVA: length(numTriangles) FREE)\
		nocopy(wVB: length(numTriangles) FREE)\
		nocopy(wVC: length(numTriangles) FREE)\
		nocopy(dArray: length(numTriangles) FREE)\
		nocopy(vertsX: length(numVerts) FREE)\
		nocopy(vertsY: length(numVerts) FREE)\
		nocopy(vertsZ: length(numVerts) FREE)\
		nocopy(velsX: length(numVerts) FREE)\
		nocopy(velsY: length(numVerts) FREE)\
		nocopy(velsZ: length(numVerts) FREE)\
		nocopy(forceX: length(numVerts) FREE)\
		nocopy(forceY: length(numVerts) FREE)\
		nocopy(forceZ: length(numVerts) FREE)\
		nocopy(indicies: length(numTriangles*3) FREE)
	#endif
   delete [] wUA ;
   delete [] wUB ;
   delete [] wUC ;
   delete [] wVA ;
   delete [] wVB ;
   delete [] wVC ;
   delete [] vertsX;
   delete [] vertsY;
   delete [] vertsZ;
   delete [] velsX;
   delete [] velsY;
   delete [] velsZ;
   delete [] indicies;
   delete [] forceX;
   delete [] forceY;
   delete [] forceZ;
   delete [] dArray;
}
