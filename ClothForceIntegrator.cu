#include "hip/hip_runtime.h"

#include "ClothForceIntegrator.h"
#include "utils.h"
#include <algorithm>
#include <iostream>

#define YOUNG_MOD 1000.0 //N/m
#define POISSON_COEFF 0.85
#define MASS 600.0 //kg/m
#define GRAVITY -4.5
#define DAMPN  0.05
#define COLLISIONSTR 20.0

#define MIN_STEP 0.0005 //Seconds

#define MAX_THREADS 1024
#define THREADS_PER_BLOCK MAX_THREADS

const double YoungPoissonMatrixScalar = YOUNG_MOD/(1 - POISSON_COEFF * POISSON_COEFF);

struct Vector
{
   double x;
   double y;
   double z;
};

struct Vector sumPoint(Vector & a, Vector & b, Vector & c,
						  double  wUA, double  wUB, double  wUC)
{

   struct Vector point;
   point.x = a.x * wUA + b.x * wUB + c.x * wUC;
   point.y = a.y * wUA + b.y * wUB + c.y * wUC;
   point.z = a.z * wUA + b.z * wUB + c.z * wUC;
   return point;
}

Vector calculateForce(Vector U, Vector V,
							 double rUJ, double rVJ, double d)
{
   double euu = 0.5 * (U.x * U.x + U.y * U.y + U.z * U.z -1);
   double evv = 0.5 * (V.x * V.x + V.y * V.y + V.z * V.z -1);
   //EUV is 0.5* (UTV + VTU), should be 2 UtV
   double euv =  U.x * V.x + U.y * V.y + U.z * V.z;

   Vector sigmas;
   sigmas.x = euu + POISSON_COEFF * evv ;
   sigmas.y = POISSON_COEFF*evv + euu ;
   sigmas.z = euv * (1 - POISSON_COEFF) / 2 ;

   sigmas.x *= YoungPoissonMatrixScalar;
   sigmas.y *= YoungPoissonMatrixScalar;
   sigmas.z *= YoungPoissonMatrixScalar;
   Vector force;
   force.x = -fabs(d)/2 *(
			   sigmas.x * rUJ * U.x +
			   sigmas.y * rVJ * V.x +
			   sigmas.z * (rUJ * V.x + rVJ * U.x));

   force.y = -fabs(d)/2 * (
			   sigmas.x * rUJ * U.y +
			   sigmas.y * rVJ * V.y +
			   sigmas.z * (rUJ * V.y + rVJ * U.y));

   force.z = -fabs(d)/2 * (
			   sigmas.x * rUJ * U.z +
			   sigmas.y * rVJ * V.z +
			   sigmas.z * (rUJ * V.z + rVJ * U.z)) ;
   return force;
}

/**
 * Setup triangle weights organized by indicies
 */
inline void ClothForceIntegrator::caluclateTriangleWeights(std::vector<float> & weights,std::vector<int> & inds)
{
   /**
	* Iterate through each triangle and calcluate a weight grouping
	*/
   for(int i = 0; i < numTriangles*3; i+=3)
   {
     floatVec2D a, b, c;
     a.x = weights[2*inds[i]];
     a.y = weights[2*inds[i]+1];
     b.x = weights[2*inds[i+1]];
     b.y = weights[2*inds[i+1]+1];
     c.x = weights[2*inds[i+2]];
     c.y = weights[2*inds[i+2]+1];
	  //create and set weights
	  float d = a.x * (b.y - c.y) + b.x * (c.y - a.y) + c.x * (a.y - b.y);
	  float recrip = 1.0 /  d;
	  dArray[i/3] = d;
	  wUA[i/3] = (b.y - c.y) * recrip;
	  wVA[i/3] = (c.x - b.x) * recrip;
	  wUB[i/3] = (c.y - a.y) * recrip;
	  wVB[i/3] = (a.x - c.x) * recrip;
	  wUC[i/3] = (a.y - b.y) * recrip;
	  wVC[i/3] = (b.x - a.x) * recrip;
   }
}

inline double lengthSq(Vector v)
{
   return v.x * v.x + v.y * v.y + v.z * v.z;
}

/**
 * Create the initial arrays, and organize data in SOA structure, push to the Phi.
 * @param cloth [description]
 */
 /**
   OrigIndicies:
   [a b c] [a b c]
   vertices:
   [x y z] [ x y z] [x y z] */
ClothForceIntegrator::ClothForceIntegrator(std::vector<int>  orig_indices, std::vector<float>  vertices, std::vector<float>  weights):
	numTriangles(orig_indices.size()/3),
	numVerts(vertices.size()/3),
	numIndicies(orig_indices.size())

{
   /**
	* Replace with __mm_malloc later
	*/
   
   std::cout << "numVerts: " << numVerts << std::endl;
   
   indicies = new int[orig_indices.size()];
   wUA = new double[numTriangles];
   wUB = new double[numTriangles];
   wUC = new double[numTriangles];
   wVA = new double[numTriangles];
   wVB = new double[numTriangles];
   wVC = new double[numTriangles];
   dArray = new double[numTriangles];

   vertsX = new double[numVerts];
   vertsY = new double[numVerts];
   vertsZ = new double[numVerts];
   velsX = new double[numVerts];
   velsY = new double[numVerts];
   velsZ = new double[numVerts];

   forceX = new double[numVerts];
   forceY = new double[numVerts];
   forceZ = new double[numVerts];
   
   expandedForceX = new double[numTriangles*3];
   expandedForceY = new double[numTriangles*3];
   expandedForceZ = new double[numTriangles*3];

   for(int i = 0; i < numVerts; i++)
   {
	  vertsX[i] = vertices[3*i];
	  vertsY[i] = vertices[3*i+1];
	  vertsZ[i] = vertices[3*i+2];
   }

   std::fill(velsX, velsX + numVerts, 0);
   std::fill(velsY, velsY + numVerts, 0);
   std::fill(velsZ, velsZ + numVerts, 0);

   std::fill(forceX, forceX + numVerts, 0);
   std::fill(forceY, forceY + numVerts, 0);
   std::fill(forceZ, forceZ + numVerts, 0);

   for(int i = 0; i < numIndicies; i++)
   {
      indicies[i] = orig_indices[i];
   }
   caluclateTriangleWeights(weights,orig_indices);
   
   
   counts = new unsigned int[numVerts];
   locs = new unsigned int[numVerts];
   outIdx = new unsigned int[numTriangles*3];
   
   for(int i = 0; i < numVerts; i++) { 	// init counts
      counts[i] = 0;
   }
   
   for(int i = 0; i < numIndicies; i++) {
      counts[indicies[i]]++;
   }
   
   std::cout << "init'd counts" << std::endl;
   
   locs[0] = 0;   	// init locs
   for(int i = 1; i < numVerts; i++) {
      locs[i] = counts[i - 1] + locs[i - 1];
   }
   
   unsigned int *tempCounts = new unsigned int[numVerts];   	// init outIdx
   for(int i = 0; i < numVerts; i++) {
      tempCounts[i] = 0;
   }
   
   for(int i = 0; i < numIndicies; i++) {
      int vertex = indicies[i];
      outIdx[i] = locs[vertex] + tempCounts[vertex];
      tempCounts[vertex]++;
   }
   
   std::cout << "dumping counts: ";
   for (int i = 0; i < numVerts; i++) {
      std::cout << counts[i] << " ";
   }
   std::cout << std::endl;
   
   std::cout << "dumping locs: ";
   for (int i = 0; i < numVerts; i++) {
      std::cout << locs[i] << " ";
   }
   std::cout << std::endl;
   
   std::cout << "dumping outIdx: ";
   for (int i = 0; i < numIndicies; i++) {
      std::cout << outIdx[i] << " ";
   }
   std::cout << std::endl;
   
    std::cout << "dumping indicies: ";
   for (int i = 0; i < numIndicies; i++) {
      std::cout << indicies[i] << " ";
   }
   std::cout << std::endl;
   
   std::cout << "ran init without dying" << std::endl;
//   exit(EXIT_FAILURE);

   const size_t vertsSize = numVerts * sizeof(double);

   checkCudaErrors(hipMalloc(&d_vertsX, vertsSize));    // init vertsXYZ on device
   checkCudaErrors(hipMalloc(&d_vertsY, vertsSize));
   checkCudaErrors(hipMalloc(&d_vertsZ, vertsSize));
   
   
   
   checkCudaErrors(hipMalloc(&d_velsX, vertsSize));  // init velsXYZ on device
   checkCudaErrors(hipMalloc(&d_velsY, vertsSize));
   checkCudaErrors(hipMalloc(&d_velsZ, vertsSize));

   const size_t indiciesSize = numIndicies * sizeof(int);

   // init d_indicies on device
   checkCudaErrors(hipMalloc(&d_indicies, indiciesSize));
   checkCudaErrors(hipMemcpy(d_indicies, indicies, indiciesSize, hipMemcpyHostToDevice));

   // init weights on device
   const size_t weightsSize = numTriangles * sizeof(double);

   checkCudaErrors(hipMalloc(&d_wUA, weightsSize)); 
   checkCudaErrors(hipMalloc(&d_wUB, weightsSize));
   checkCudaErrors(hipMalloc(&d_wUC, weightsSize));
   checkCudaErrors(hipMalloc(&d_wVA, weightsSize));
   checkCudaErrors(hipMalloc(&d_wVB, weightsSize));
   checkCudaErrors(hipMalloc(&d_wVC, weightsSize));
   checkCudaErrors(hipMalloc(&d_dArray, weightsSize));

   checkCudaErrors(hipMemcpy(d_wUA, wUA, weightsSize, hipMemcpyHostToDevice)); 
   checkCudaErrors(hipMemcpy(d_wUB, wUB, weightsSize, hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_wUC, wUC, weightsSize, hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_wVA, wVA, weightsSize, hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_wVB, wVB, weightsSize, hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_wVC, wVC, weightsSize, hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_dArray, dArray, weightsSize, hipMemcpyHostToDevice));

   // inid d_outIdx on device
   const size_t outIdxSize = numTriangles * 3 * sizeof(unsigned int);

   checkCudaErrors(hipMalloc(&d_outIdx, outIdxSize));
   checkCudaErrors(hipMemcpy(d_outIdx, outIdx, outIdxSize, hipMemcpyHostToDevice));

   // allocate d_expandedForceXYZ
   const size_t expandedForceSize = numTriangles * 3 * sizeof(double);

   checkCudaErrors(hipMalloc(&d_expandedForceX, expandedForceSize));
   checkCudaErrors(hipMalloc(&d_expandedForceY, expandedForceSize));
   checkCudaErrors(hipMalloc(&d_expandedForceZ, expandedForceSize));

   // DEBUG

   // verify init of d_indicies
   checkCudaErrors(hipMemcpy(indicies, d_indicies, indiciesSize, hipMemcpyDeviceToHost));

   // verify init of weights on device
   checkCudaErrors(hipMemcpy(wUA, d_wUA, weightsSize, hipMemcpyDeviceToHost)); 
   checkCudaErrors(hipMemcpy(wUB, d_wUB, weightsSize, hipMemcpyDeviceToHost));
   checkCudaErrors(hipMemcpy(wUC, d_wUC, weightsSize, hipMemcpyDeviceToHost));
   checkCudaErrors(hipMemcpy(wVA, d_wVA, weightsSize, hipMemcpyDeviceToHost));
   checkCudaErrors(hipMemcpy(wVB, d_wVB, weightsSize, hipMemcpyDeviceToHost));
   checkCudaErrors(hipMemcpy(wVC, d_wVC, weightsSize, hipMemcpyDeviceToHost));
   checkCudaErrors(hipMemcpy(dArray, d_dArray, weightsSize, hipMemcpyDeviceToHost));

   // verify init of d_outIdx
   checkCudaErrors(hipMemcpy(outIdx, d_outIdx, outIdxSize, hipMemcpyDeviceToHost));
}

#define FABS(X) ((X) < 0 ? (-(X)) : (X))

__global__
void findExpandedForce(size_t numTriangles,
                       double *vertsX, double *vertsY, double *vertsZ,
                       double *velsX, double *velsY, double *velsZ,
                       int *indicies,
                       double * wUA, double * wUB, double * wUC, double * wVA, double * wVB, double * wVC, double * dArray,
                       double * expandedForceX, double * expandedForceY, double * expandedForceZ,
                       unsigned int * outIdx) {
   size_t triNo = blockIdx.x * blockDim.x + threadIdx.x;

   if (triNo < numTriangles) {
      Vector A, B, C, vA, vB, vC;

      A.x = vertsX[indicies[triNo*3]];       // find positions and velocities of each point on this triangle
      B.x = vertsX[indicies[triNo*3 + 1]];
      C.x = vertsX[indicies[triNo*3 + 2]];

      A.y = vertsY[indicies[triNo*3]];
      B.y = vertsY[indicies[triNo*3 + 1]];
      C.y = vertsY[indicies[triNo*3 + 2]];

      A.z = vertsZ[indicies[triNo*3]];
      B.z = vertsZ[indicies[triNo*3 + 1]];
      C.z = vertsZ[indicies[triNo*3 + 2]];

      vA.x = velsX[indicies[triNo*3]];
      vB.x = velsX[indicies[triNo*3 + 1]];
      vC.x = velsX[indicies[triNo*3 + 2]];

      vA.y = velsY[indicies[triNo*3]];
      vB.y = velsY[indicies[triNo*3 + 1]];
      vC.y = velsY[indicies[triNo*3 + 2]];

      vA.z = velsZ[indicies[triNo*3]];
      vB.z = velsZ[indicies[triNo*3 + 1]];
      vC.z = velsZ[indicies[triNo*3 + 2]];


      Vector U, V;
      U.x = A.x * wUA[triNo] + B.x * wUB[triNo] + C.x * wUC[triNo];
      U.y = A.y * wUA[triNo] + B.y * wUB[triNo] + C.y * wUC[triNo];
      U.z = A.z * wUA[triNo] + B.z * wUB[triNo] + C.z * wUC[triNo];

      V.x = A.x * wVA[triNo] + B.x * wVB[triNo] + C.x * wVC[triNo];
      V.y = A.y * wVA[triNo] + B.y * wVB[triNo] + C.y * wVC[triNo];
      V.z = A.z * wVA[triNo] + B.z * wVB[triNo] + C.z * wVC[triNo];

      double d = dArray[triNo];

      Vector forceA, forceB, forceC;
      double euu, evv, euv;
      Vector sigmas;

      euu = 0.5 * (U.x * U.x + U.y * U.y + U.z * U.z -1);
      evv = 0.5 * (V.x * V.x + V.y * V.y + V.z * V.z -1);
      euv =  U.x * V.x + U.y * V.y + U.z * V.z;

      double youngPoissonMatrixScalar = YOUNG_MOD/(1 - POISSON_COEFF * POISSON_COEFF);

      sigmas.x = euu + POISSON_COEFF * evv ;
      sigmas.y = POISSON_COEFF*evv + euu ;
      sigmas.z = euv * (1 - POISSON_COEFF) / 2 ;

      sigmas.x *= youngPoissonMatrixScalar;
      sigmas.y *= youngPoissonMatrixScalar;
      sigmas.z *= youngPoissonMatrixScalar;

      if (d < 0) {
         d = -d;
      }

      forceA.x = -(d)/2 *(
               sigmas.x * wUA[triNo] * U.x +
               sigmas.y * wVA[triNo] * V.x +
               sigmas.z * (wUA[triNo] * V.x + wVA[triNo] * U.x));

      forceA.y = -(d)/2 * (
               sigmas.x * wUA[triNo] * U.y +
               sigmas.y * wVA[triNo] * V.y +
               sigmas.z * (wUA[triNo] * V.y + wVA[triNo] * U.y));

      forceA.z = -(d)/2 * (
               sigmas.x * wUA[triNo] * U.z +
               sigmas.y * wVA[triNo] * V.z +
               sigmas.z * (wUA[triNo] * V.z + wVA[triNo] * U.z));

      expandedForceX[outIdx[triNo*3]] = forceA.x - DAMPN * vA.x;
      expandedForceY[outIdx[triNo*3]] = forceA.y - DAMPN * vA.y;
      expandedForceZ[outIdx[triNo*3]] = forceA.z - DAMPN * vA.z;

      forceB.x = -(d)/2 *(
               sigmas.x * wUB[triNo] * U.x +
               sigmas.y * wVB[triNo] * V.x +
               sigmas.z * (wUB[triNo] * V.x + wVB[triNo] * U.x));

      forceB.y = -(d)/2 * (
               sigmas.x * wUB[triNo] * U.y +
               sigmas.y * wVB[triNo] * V.y +
               sigmas.z * (wUB[triNo] * V.y + wVB[triNo] * U.y));

      forceB.z = -(d)/2 * (
               sigmas.x * wUB[triNo] * U.z +
               sigmas.y * wVB[triNo] * V.z +
               sigmas.z * (wUB[triNo] * V.z + wVB[triNo] * U.z));

      forceC.x = -(d)/2 *(
               sigmas.x * wUC[triNo] * U.x +
               sigmas.y * wVC[triNo] * V.x +
               sigmas.z * (wUC[triNo] * V.x + wVC[triNo] * U.x));

      forceC.y = -(d)/2 * (
               sigmas.x * wUC[triNo] * U.y +
               sigmas.y * wVC[triNo] * V.y +
               sigmas.z * (wUC[triNo] * V.y + wVC[triNo] * U.y));

      forceC.z = -(d)/2 * (
               sigmas.x * wUC[triNo] * U.z +
               sigmas.y * wVC[triNo] * V.z +
               sigmas.z * (wUC[triNo] * V.z + wVC[triNo] * U.z));

      

      //Update second vertex
      expandedForceX[outIdx[triNo*3+1]] = forceB.x - DAMPN * vB.x;
      expandedForceY[outIdx[triNo*3+1]] = forceB.y - DAMPN * vB.y;
      expandedForceZ[outIdx[triNo*3+1]] = forceB.z - DAMPN * vB.z;

      //update third vertex
      expandedForceX[outIdx[triNo*3 + 2]] = forceC.x - DAMPN * vC.x;
      expandedForceY[outIdx[triNo*3 + 2]] = forceC.y - DAMPN * vC.y;
      expandedForceZ[outIdx[triNo*3 + 2]] = forceC.z - DAMPN * vC.z;
   }
}

void ClothForceIntegrator::step(double stepAmnt, float * outputVertices, std::vector<int> & theLockedVerts )
{
   
   const size_t vertsSize = numVerts * sizeof(double);
   
   
   
   //Calculate a force vector
   int numLockedVerts = theLockedVerts.size();
   int *lockedVerts = new int[numLockedVerts];  	  // initialize locked verts (map)
   for(int i = 0; i < theLockedVerts.size(); i++)
   {
      lockedVerts[i] = theLockedVerts[i];
   }

   for(int steps = 0; steps < (int)(stepAmnt/MIN_STEP); steps++)  	// for each step in a frame
   {
      const double dt = MIN_STEP;
      const double recipMass = 1/(MASS/numVerts);

      /* launch a thread for each triandle and write results to expanded force
      shit we will need initialized:
         d_vertsXYZ   -
         d_velsXYZ    -
         d_indicies   -
         d_(weights)  -
         d_outIdx     -
      shit we will need allocated:
         d_expandedForceX -
      */

      checkCudaErrors(hipMemcpy(d_vertsX, vertsX, vertsSize, hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(d_vertsY, vertsY, vertsSize, hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(d_vertsZ, vertsZ, vertsSize, hipMemcpyHostToDevice));

      checkCudaErrors(hipMemcpy(d_velsX, velsX, vertsSize, hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(d_velsY, velsY, vertsSize, hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(d_velsZ, velsZ, vertsSize, hipMemcpyHostToDevice));

      dim3 blocks, threads;

      blocks = dim3(numTriangles / THREADS_PER_BLOCK + 1);
      threads = dim3(THREADS_PER_BLOCK);

      findExpandedForce<<<blocks, threads>>>(numTriangles, d_vertsX, d_vertsY, d_vertsZ, d_velsX, d_velsY, d_velsZ, d_indicies, d_wUA, d_wUB, d_wUC, d_wVA, d_wVB, d_wVC, d_dArray, d_expandedForceX, d_expandedForceY, d_expandedForceZ, d_outIdx);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	   // for(int i = 0; i < numTriangles; i++)
	   // {
		  // Vector A, B, C, vA, vB, vC;

		  // A.x = vertsX[indicies[i*3]]; 	     // find positions and velocities of each point on this triangle
		  // B.x = vertsX[indicies[i*3 + 1]];
		  // C.x = vertsX[indicies[i*3 + 2]];

		  // A.y = vertsY[indicies[i*3]];
		  // B.y = vertsY[indicies[i*3 + 1]];
		  // C.y = vertsY[indicies[i*3 + 2]];

		  // A.z = vertsZ[indicies[i*3]];
		  // B.z = vertsZ[indicies[i*3 + 1]];
		  // C.z = vertsZ[indicies[i*3 + 2]];

		  // vA.x = velsX[indicies[i*3]];
		  // vB.x = velsX[indicies[i*3 + 1]];
		  // vC.x = velsX[indicies[i*3 + 2]];

		  // vA.y = velsY[indicies[i*3]];
		  // vB.y = velsY[indicies[i*3 + 1]];
		  // vC.y = velsY[indicies[i*3 + 2]];

		  // vA.z = velsZ[indicies[i*3]];
		  // vB.z = velsZ[indicies[i*3 + 1]];
		  // vC.z = velsZ[indicies[i*3 + 2]];

		  // Vector U = sumPoint(A,B,C,wUA[i],wUB[i],wUC[i]);  	// find U,V for this triangle
		  // Vector V = sumPoint(A,B,C,wVA[i],wVB[i],wVC[i]);

		  // Vector forceA = calculateForce(U,V,wUA[i],wVA[i],dArray[i]);  	// calculate force on each point in triandle
		  // Vector forceB = calculateForce(U,V,wUB[i],wVB[i],dArray[i]);
		  // Vector forceC = calculateForce(U,V,wUC[i],wVC[i],dArray[i]);

		  // //Update first vertex
		  // expandedForceX[outIdx[i*3]] = forceA.x - DAMPN * vA.x;
		  // expandedForceY[outIdx[i*3]] = forceA.y - DAMPN * vA.y;
		  // expandedForceZ[outIdx[i*3]] = forceA.z - DAMPN * vA.z;

		  // //Update second vertex
		  // expandedForceX[outIdx[i*3+1]] = forceB.x - DAMPN * vB.x;
		  // expandedForceY[outIdx[i*3+1]] = forceB.y - DAMPN * vB.y;
		  // expandedForceZ[outIdx[i*3+1]] = forceB.z - DAMPN * vB.z;

		  // //update third vertex
		  // expandedForceX[outIdx[i*3 + 2]] = forceC.x - DAMPN * vC.x;
		  // expandedForceY[outIdx[i*3 + 2]] = forceC.y - DAMPN * vC.y;
		  // expandedForceZ[outIdx[i*3 + 2]] = forceC.z - DAMPN * vC.z;
	   // }

      const size_t expandedForceSize = numTriangles * 3 * sizeof(double); // FIXME: this should be const class member

      checkCudaErrors(hipMemcpy(expandedForceX, d_expandedForceX, expandedForceSize, hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy(expandedForceY, d_expandedForceY, expandedForceSize, hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy(expandedForceZ, d_expandedForceZ, expandedForceSize, hipMemcpyDeviceToHost));

      for(int i = 0; i < numVerts; i++) { // calc forceXYZ with expandedForceXYZ
         int locMax = (i+1 == numVerts)? numVerts :  locs[i+1];
         for(int j = locs[i]; j < locMax; j++  ){
            forceX[i] += expandedForceX[j];
            forceY[i] += expandedForceY[j];
            forceZ[i] += expandedForceZ[j];
         }
      }

	   for(int i = 0; i < numVerts; i++)
	   {
		  forceY[i] += GRAVITY;
	   }
      
	   for(int i = 0; i < numLockedVerts; ++i)
	   {
		  forceX[lockedVerts[i]]=0;
		  forceY[lockedVerts[i]]=0;
		  forceZ[lockedVerts[i]]=0;
	   }

	   for(int i = 0; i < numVerts; i++)
	   {
		  velsX[i]  += forceX[i] * dt * recipMass;
		  vertsX[i] += velsX[i] * dt;

		  velsY[i]  += forceY[i] * dt * recipMass;
		  vertsY[i] += velsY[i] * dt;

		  velsZ[i]  += forceZ[i] * dt * recipMass;
		  vertsZ[i] += velsZ[i] * dt;
	   }

      for(int i = 0; i < numVerts; i++) {
         forceX[i] = forceY[i] = forceZ[i] = 0;
      }
   }
   delete [] lockedVerts;
   /**
	* Set final vertex positions
	*/
   for(int i = 0; i < numVerts; i++)
   {
	  outputVertices[i*3] = (float)vertsX[i];
	  outputVertices[i*3+1] = (float)vertsY[i];
	  outputVertices[i*3+2] = (float)vertsZ[i];  
   }
}

ClothForceIntegrator::~ClothForceIntegrator()
{
   delete [] wUA ;
   delete [] wUB ;
   delete [] wUC ;
   delete [] wVA ;
   delete [] wVB ;
   delete [] wVC ;
   delete [] vertsX;
   delete [] vertsY;
   delete [] vertsZ;
   delete [] velsX;
   delete [] velsY;
   delete [] velsZ;
   delete [] indicies;
   delete [] forceX;
   delete [] forceY;
   delete [] forceZ;
   delete [] dArray;
}
